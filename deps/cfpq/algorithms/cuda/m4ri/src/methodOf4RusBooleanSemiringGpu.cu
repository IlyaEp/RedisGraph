#include "hip/hip_runtime.h"

#include "methodOf4RusBooleanSemiringGpu.h"
#include <hip/hip_runtime_api.h>
#include <>
#include <device_types.h>

#define lsb(i) ((i) & -(i)) // return least significant bit

namespace gpu_m4ri {

__device__ bool is_changed_matrix = false;

__device__ size_t result = 0;

__global__ void sum_impl(const TYPE *d_data, size_t size) {
  if (blockIdx.x > 0)
    return;

  auto threads_per_block = blockDim.x;

  const auto threadId = threadIdx.x;
  extern __shared__ size_t s_data[];

  size_t thread_result = 0;
  for (size_t i = threadId; i < size; i += threads_per_block) {
    thread_result += __popc(d_data[i]);
  }

  s_data[threadId] = thread_result;
  __syncthreads();

  if (threadId == 0) {
    result = 0;
    for (size_t i = 0; i < threads_per_block; i++) {
      result += s_data[i];
    }
  }
}

size_t sum(const TYPE *d_data, size_t size) {
  size_t res;
  sum_impl<<<1, 512, 512 * sizeof(size_t)>>>(d_data, size);
  hipMemcpyFromSymbol(&res, HIP_SYMBOL(result), sizeof(size_t), 0, hipMemcpyDeviceToHost);
  return res;
}

// return the next number with the same number of bits
__device__ int snoob(int i) {
  int least = lsb(i);
  int ripple = i + least;
  return (((ripple ^ i) >> 2) / least) | ripple;
}

__global__ void make_table_kernel_semiring(TYPE *B, TYPE **lookup_tables,
                                           int cols, int rows, int tables_num,
                                           int real_cols, int offset) {
  int x_col = blockIdx.x * BLOCK_SIZE_COL + threadIdx.x;
  int y_row = (blockIdx.y * BLOCK_SIZE_ROW + threadIdx.y) * K;
  int twokey = (1 << K);
  int i;
  int least, rest;

  if (x_col >= cols || y_row >= rows) {
    // if thread out of current computed part of the table then return
    return;
  }

  // pointer to calculated  table  Br
  TYPE *T = lookup_tables[blockIdx.y * BLOCK_SIZE_ROW + threadIdx.y];

  T[x_col] = 0; // row with 000000000...

// fill when table when 1 bit
#pragma unroll
  for (int j = 0; j < K; j++) {
    i = 1 << (j);
    T[i * cols + x_col] = B[(y_row + j) * real_cols + x_col + offset];
  }

#pragma unroll
  for (int h = 2; h <= K; h++) {
    // fill table elems with h bits in index and <2^K
    // idea taken from Peter Schäfer implementation for m4ri library
    i = (1 << h) - 1;
    for (; i < twokey; i = snoob(i)) {
      least = lsb(i);
      rest = i - least;
      T[i * cols + x_col] = T[least * cols + x_col] | T[rest * cols + x_col];
    }
  }
}

__device__ int get_actual_key(TYPE composite_key, int j) {
  return (0xFF) & (composite_key >> (8 * j));
}

__global__ void m4ri_mul_kernel_semiring(TYPE *A, TYPE *C, TYPE **lookup_tables,
                                         int rows, int cols, int cols_table,
                                         int offset) {
  __shared__ TYPE local_A[BLOCK_SIZE_ROW][BLOCK_SIZE_COL];
  int col_x = threadIdx.x + blockIdx.x * BLOCK_SIZE_COL + offset;
  int row_y = threadIdx.y + blockIdx.y * BLOCK_SIZE_ROW;
  int col_in_T = threadIdx.x + blockIdx.x * BLOCK_SIZE_COL;
  int full_steps = cols / BLOCK_SIZE_COL;
  int small_step = cols % BLOCK_SIZE_COL;
  TYPE *T;
  TYPE composite_key;
  int actual_key;
  TYPE old_c;

  if (col_x < cols && col_in_T < cols_table && row_y < rows) {
    // if not out
    old_c = C[row_y * cols + col_x];
  } else {
    old_c = 0;
  }

  TYPE tmp;
  TYPE value = 0;

#pragma unroll
  for (int i = 0; i < full_steps; i++) {
    // all complete runs
    tmp = __brev(A[row_y * cols + threadIdx.x + i * BLOCK_SIZE_COL]); // reverse
    local_A[threadIdx.y][threadIdx.x] = tmp;
    __syncthreads();

    for (int t = 0; t < BLOCK_SIZE_COL; t++) {
      composite_key = local_A[threadIdx.y][t];
      for (int j = 0; j < 4; j++) {
        T = lookup_tables[BLOCK_SIZE_COL * i * 4 + t * 4 + j];
        actual_key = get_actual_key(composite_key, j);
        value |= T[actual_key * cols_table + col_in_T];
      }
    }
  }
  __syncthreads();
  if (small_step) {
    int cur_step = full_steps;
    if (threadIdx.x + cur_step * BLOCK_SIZE_COL < cols && row_y < rows) {
      tmp = __brev(
          A[row_y * cols + threadIdx.x + cur_step * BLOCK_SIZE_COL]); // reverse
      local_A[threadIdx.y][threadIdx.x] = tmp;
    }
    __syncthreads();

    if (col_x >= cols || col_in_T >= cols_table || row_y >= rows) {
      // threads that out of current part of C contributed to all threads(load
      // keys) and can return
      return;
    }

    for (int t = 0; t < small_step; t++) {
      composite_key = local_A[threadIdx.y][t];
      for (int j = 0; j < 4; j++) {
        T = lookup_tables[cur_step * BLOCK_SIZE_COL * 4 + t * 4 + j];
        actual_key = get_actual_key(composite_key, j);
        value |= T[actual_key * cols_table + col_in_T];
      }
    }
  }
  value = value | old_c;

  if (is_changed_matrix == false && value != old_c) {
    is_changed_matrix = true;
  }

  if (col_x < cols && row_y < rows && col_in_T < cols_table && value != old_c) {
    C[row_y * cols + col_x] = old_c | value;
  }
}

int wrapper_method_of_4rus_bool_semiring(TYPE *a, TYPE *b, TYPE *c,
                                         Tables &tables, int rows, int cols) {
  int is_c_changed = false;
  hipMemcpyToSymbol(HIP_SYMBOL(is_changed_matrix), &is_c_changed, sizeof(bool), 0,
                     hipMemcpyHostToDevice);

  // setup configuration for table kernel
  dim3 dimBlock_table_kernel(BLOCK_SIZE_COL, BLOCK_SIZE_ROW);

  dim3 dimGrid_table_n((tables.cols_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL,
                       (rows + BLOCK_SIZE_ROW * K - 1) / (BLOCK_SIZE_ROW * K));

  dim3 dimGrid_table_last(
      (tables.cols_last + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL,
      (rows + BLOCK_SIZE_ROW * K - 1) / (BLOCK_SIZE_ROW * K));

  // setup configuration for mul kernel
  dim3 dimBlock_m4ri(BLOCK_SIZE_COL, BLOCK_SIZE_ROW);

  dim3 dimGrid_m4ri_n((tables.cols_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL,
                      ((rows + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW));

  dim3 dimGrid_m4ri_last((tables.cols_last + BLOCK_SIZE_COL - 1) /
                             BLOCK_SIZE_COL,
                         ((rows + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW));

  for (int i = 0; i < tables.num_launches; i++) {
    make_table_kernel_semiring<<<dimGrid_table_n, dimBlock_table_kernel>>>(
        b, tables.table_n, tables.cols_n, rows, tables.num_tables, cols,
        i * tables.cols_n);
    hipDeviceSynchronize();
    m4ri_mul_kernel_semiring<<<dimGrid_m4ri_n, dimBlock_m4ri>>>(
        a, c, tables.table_n, rows, cols, tables.cols_n, i * tables.cols_n);
    hipDeviceSynchronize();
  }

  if (tables.cols_last != 0) {
    make_table_kernel_semiring<<<dimGrid_table_last, dimBlock_table_kernel>>>(
        b, tables.table_last, tables.cols_last, rows, tables.num_tables, cols,
        tables.num_launches * tables.cols_n);
    hipDeviceSynchronize();
    m4ri_mul_kernel_semiring<<<dimGrid_m4ri_last, dimBlock_m4ri>>>(
        a, c, tables.table_last, rows, cols, tables.cols_last,
        tables.num_launches * tables.cols_n);
    hipDeviceSynchronize();
  }

  hipMemcpyFromSymbol(&is_c_changed, HIP_SYMBOL(is_changed_matrix), sizeof(bool), 0,
                       hipMemcpyDeviceToHost);
  return is_c_changed;
}
} // namespace gpu_m4ri
