#include "gpuMemoryManagement.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <iostream>

namespace gpu_m4ri {

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

TYPE **allocate_tables(int num_tables, int num_rows, int num_cols) {
  TYPE **d_ppcPtr, *d_pcPtr;
  gpuErrchk(hipMalloc(&d_ppcPtr, sizeof(TYPE *) * num_tables))

      for (int i = 0; i < num_tables; i++) {
    gpuErrchk(hipMalloc(&d_pcPtr, sizeof(TYPE) * num_rows * num_cols))
        gpuErrchk(hipMemset(d_pcPtr, 0, sizeof(TYPE) * num_rows * num_cols))
            gpuErrchk(hipMemcpy(&d_ppcPtr[i], &d_pcPtr, sizeof(TYPE *),
                                 hipMemcpyHostToDevice))
  }
  return d_ppcPtr;
}

void delete_tables(TYPE **tables, int num_tables) {
  TYPE **someHost;
  gpuErrchk(hipHostMalloc((void **)&someHost, sizeof(TYPE *) * num_tables))
      gpuErrchk(hipMemcpy(someHost, tables, num_tables * sizeof(TYPE *),
                           hipMemcpyDeviceToHost))

          for (int i = 0; i < num_tables; i++){gpuErrchk(
              hipFree((void *)someHost[i]))} gpuErrchk(hipFree(tables))
              gpuErrchk(hipHostFree(someHost))
}

TYPE *allocate_matrix_host(int rows, int cols) {
  TYPE *matrix;
  gpuErrchk(hipHostMalloc((void **)&matrix,
                           sizeof(TYPE) * rows * cols)) return matrix;
}

void delete_matrix_host(TYPE *matrix){gpuErrchk(hipHostFree(matrix))}

TYPE *allocate_matrix_device(int rows, int cols) {
  TYPE *matrix;
  gpuErrchk(
      hipMalloc((void **)&matrix, sizeof(TYPE) * rows * cols)) return matrix;
}

void delete_matrix_device(TYPE *matrix) { gpuErrchk(hipFree(matrix)) }

void copy_device_to_host_sync(TYPE *device, TYPE *host, int elems) {
  gpuErrchk(
      hipMemcpy(host, device, sizeof(TYPE) * elems, hipMemcpyDeviceToHost))
}

void copy_host_to_device_sync(TYPE *host, TYPE *device, int elems) {
  gpuErrchk(
      hipMemcpy(device, host, sizeof(TYPE) * elems, hipMemcpyHostToDevice))
}

void copy_device_to_device_sync(TYPE *src, TYPE *dst, int elems) {
  gpuErrchk(
      hipMemcpy(dst, src, sizeof(TYPE) * elems, hipMemcpyDeviceToDevice))
}

void copy_device_to_host_async(TYPE *device, TYPE *host, int elems) {
  gpuErrchk(hipMemcpyAsync(host, device, sizeof(TYPE) * elems,
                            hipMemcpyDeviceToHost))
}

void copy_host_to_device_async(TYPE *host, TYPE *device, int elems) {
  gpuErrchk(hipMemcpyAsync(device, host, sizeof(TYPE) * elems,
                            hipMemcpyHostToDevice))
}

void synchronize_with_gpu() { hipDeviceSynchronize(); }

} // namespace gpu_m4ri
